
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
	printf("Hello Cuda!\n");
}

int main() {

	hello_cuda<<<2,1>>>();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

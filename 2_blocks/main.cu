
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
	printf("Hello Cuda!\n");
}

int main() {

	dim3 block(4);
	dim3 grid(8);

	hello_cuda<<<grid, block>>>();;
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
